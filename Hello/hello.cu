
#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(char *ad, int *bd) 
{
  int tidx = blockIdx.x;
  int tidy = blockIdx.y;
  int tidz = threadIdx.x;
  printf("TID = <%d,%d,%d>\n", tidx, tidy, tidz);
}
 
int main()
{
  char a[N] = "Hello \0\0\0\0\0\0";
  int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);
 
  printf("%s", a);
 
  hipMalloc( (void**)&ad, csize ); 
  hipMalloc( (void**)&bd, isize ); 
  hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
  hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
  
  dim3 dimGrid( 25, 25 );
  dim3 dimBlock( 10, 10 );
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
  hipFree( ad );
  hipFree( bd );

  printf("%s\n", a);
  return EXIT_SUCCESS;
}
